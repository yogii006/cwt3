
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    printf("Hello CUDA from GPU!\n");
}

int main()
{
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    printf("cuda checking centre");
    return 0;
}
